
#include <hip/hip_runtime.h>
#include<stdio.h>
#define CHECK(res) if(res!=hipSuccess){exit(-1);}
const int width=5;
const int height=22;
const int size=width*height*sizeof(int );
//const int size=sizeof(int)*width;

/*__global__ void kerneltest(int **b,size_t pitch)
{
	printf("(%d,%d)\n",threadIdx.x,threadIdx.y);
	int *c=(int *)((char *)b+threadIdx.x*pitch);
	printf("%d, ",c[threadIdx.y]);
}

int main(int argc,char **argv)
{
	int i,j;
	int a[height][width];
	int c[height][width];
	int **b;
	size_t pitch;
	cudaError_t res;
	for(i=0;i<height;i++)
	{
		for(j=0;j<width;j++)
		{
			a[i][j]=j+i*width;
			c[i][j]=0;
			printf("%d  ",a[i][j]);
		}
		printf("\n");
	}
	res=cudaMallocPitch((void **)&b,&pitch,size,height);CHECK(res);printf("1");
	res=cudaMemcpy2D(b,pitch,a,size,size,height,cudaMemcpyHostToDevice);CHECK(res);printf("2");
	dim3 threads(5,10);
	dim3 blocks(1,1);
	kerneltest<<<blocks,threads>>>(b,pitch);
	printf("3");
	res=cudaMemcpy2D(c,size,b,pitch,size,height,cudaMemcpyDeviceToHost);CHECK(res);printf("4\n");
	for(i=0;i<height;i++)
	{
		for(j=0;j<width;j++)
		{
			printf("%d  ",c[i][j]);
		}
		printf("\n");
	}
	cudaFree(b);
	return 0;
}
*/
__global__ void testkernel(int **b,int dheight,int dwidth,int dmul)
{
	//printf("%d\n",add[threadIdx.x+blockIdx.x*blockDim.x]);
	/*if(threadIdx.x+blockIdx.x*blockDim.x<dwidth){
	int i,idx,idy;
	int num=0;
	for(i=0;i<dheight;i++)
	{
		idx=(threadIdx.x+i*dwidth)/blockDim.x;
		idy=(threadIdx.x+i*dwidth)%blockDim.x;
		num+=add[idy+idx*blockDim.x];
	}
	printf("%d  ",num);}*/
	int i,j,num;
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	int idy;
	printf("(%d,%d)\n",threadIdx.x,idx);
	int *add=(int *)((int *)b);//,pointarr[threadIdx.y][threadIdx.x]);
	for(i=0;i<dmul;i++)
	{
		idy=dmul*idx+i;
		num=0;
		if(idy<dheight){

			for(j=0;j<dwidth;j++)
			{
				num+=add[idy*dwidth+j];
			}
			printf("(%d,%d):%d\n",idx,idy,num);
		}
	}
}

int main()
{
	int a[height][width];
	int c[height][width];
	int **b;
	int thread_size=10;
	int block_size=1;
	int mul;
	mul=(height/(thread_size*block_size))+1;
	for(int i=0;i<height;i++)
	{
		for(int j=0;j<width;j++)
		{
			a[i][j]=j+i*width;
			c[i][j]=0;
			printf("%d  ",a[i][j]);
		}
		printf("\n");
	}
	hipMalloc((void **)&b,size);
	hipMemcpy(b,a,size,hipMemcpyHostToDevice);
	dim3 threads(thread_size,1);
	dim3 blocks(block_size,1);
	testkernel<<<blocks,threads>>>(b,height,width,mul);
	hipDeviceSynchronize();
	hipMemcpy(c,b,size,hipMemcpyDeviceToHost);
	for(int i=0;i<height;i++)
	{
		for(int j=0;j<width;j++)
		{
			printf("%d  ",c[i][j]);
		}
		printf("\n");
	}
	return 0;
}
