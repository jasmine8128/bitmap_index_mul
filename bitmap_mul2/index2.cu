
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define CHECK(res) if (res!=hipSuccess){exit(-1);}	//check if success
const int height=10;		//the kinds of attributes(>=)
const int width=100;		//the kinds of datas(>=)
const int kinds=30;			//the kinds of types(>=)
const int bit_size=width*height*sizeof(int );	//the size of bitat[][]
const int index_size=width*sizeof(int );
unsigned int bit[height][width];
unsigned int bitat[width][height];	//add 0-Fill data
int key[height][kinds];
int offset[height][kinds];
int index_bit[width];
int index_long[1];
int attr_size;				//the kinds of attributes(=)
int attr_total;				//the kinds of datas/31 (=)
unsigned int bin_31=0x80000000;
FILE *fp;
char str[33];
hipError_t res;

void my_itoa(int num,char *strr,int bin2)	//change num(decimal) into strr(binary)
{
    int i;
    int b=0x00000001;
    for(i=31;i>=0;i--)
    {
        if(num&b)
            strr[i]='1';
        else
            strr[i]='0';
        num=num>>1;
    }
    strr[32]='\0';
}

void get_attr_size()	//get attr_size
{
	fp=fopen("outputm.txt","r");
	char c;
	attr_size=0;
	while((c=fgetc(fp))!=EOF)
	{
		if(c=='[')
			attr_size++;

	}
	attr_size=attr_size/2;
	fclose(fp);
}
void get_bitmap()  //get bitmap,key and offset from file
{
	fp=fopen("outputm.txt","r");
	int i,j,k,offs;
	char init;
	i=0;j=0;k=0;
	fscanf(fp,"%d",&bit[i][j]);j++;
	while((init=fgetc(fp))!=EOF)
	{
		if(init=='[')
		{
			fscanf(fp,"%d",&offs);
			while(fgetc(fp)!=']')
			{
				key[i][k]=offs;k++;
				fscanf(fp,"%d",&offs);
			}
			key[i][k]=offs;
			while(fgetc(fp)!='[');k=0;
			fscanf(fp,"%d",&offs);
			while(fgetc(fp)!=']')
			{
				offset[i][k]=offs;k++;
				fscanf(fp,"%d",&offs);
			}
			offset[i][k]=offs;
			i++;j=0;k=0;
		}
		else{
			fscanf(fp,"%d",&bit[i][j]);
			j++;
		}
	}
}

void get_total()
{
	int i,tsize,tlie;
	attr_total=0;
	tsize=key[0][0];
	tlie=offset[0][0];
	for(i=0;i<tsize;i++)
	{
		attr_total++;
		if(bit[0][tlie+i]<=bin_31)
			attr_total=attr_total+bit[0][tlie+i]-1;
	}
	printf("attr_total:%d\n",attr_total);

}
void get_attr() //get attr from screen,store them in the bitat[][]
{
	int i,j,k,attr;
	int size[height];
	int lie[height];
	int local;
	index_long[0]=0;
	for(i=0;i<attr_total;i++)
	{
		for(j=0;j<attr_size;j++)
			bitat[i][j]=0xffffffff;
	}
	for(i=0;i<attr_size;i++)
	{
		printf("Please input the attribute you choose(if not,input -1):\n");
		scanf("%d",&attr);
		if(attr==-1)
		{
			size[i]=0;
			lie[i]=0;
		}
		else{
			size[i]=key[i][attr];  //find key and offset
			lie[i]=offset[i][attr];
		}
	}
	for(i=0;i<attr_size;i++)		//store bitmap in the bitat[][]
	{
		local=-1;
		for(j=0;j<size[i];j++)
		{
			local+=1;
			if(bit[i][lie[i]+j]>bin_31)	//not 0-Fill
			{
				bitat[local][i]=bit[i][lie[i]+j];
			}
			else						//0-Fill
			{
				for(k=0;k<bit[i][lie[i]+j];k++)
				{
					bitat[local+k][i]=0;
				}
				local=local+bit[i][lie[i]+j]-1;
			}
		}
	}

}
__device__ void d_itoa(int num,char *strr)	//device change num(decimal) into strr(binary)
{
    int i;
    int b=0x00000001;
    for(i=31;i>=0;i--)
    {
        if(num&b)
            strr[i]='1';
        else
            strr[i]='0';
        num=num>>1;
    }
    strr[32]='\0';
}
__global__ void kernel_index_bitmap(unsigned int **dbit,int *dindex_bit,int *dindex_long,int dtotal,int dsize,int dheight,int dmul)
{
	int i,j,k,addr;
	char strr[33];
	unsigned int num;
	int idx=threadIdx.x+blockIdx.x*blockDim.x;
	int idy;
	int *add=(int *)((int *)dbit);		//the address of the bitat[][]
	for(i=0;i<dmul;i++)
	{
		idy=dmul*idx+i;
		num=0xffffffff;					//num=32 bits of '1'
		if(idy<dtotal)
		{
			for(j=0;j<dsize;j++)
			{
				num&=add[idy*dheight+j];
				printf("(%d,%d):%d\n",idy,idy*dheight+j,add[idy*dheight+j]);
			}
			printf("num:(%d,%d):%d\n",idx,idy*dheight+j,num);
			d_itoa(num,strr);
			printf("%d:%s\n",idy,strr);
			for(j=1;j<32;j++)
			{
				if(strr[j]=='1')
				{
					addr=idy*31+j;
					printf("attr:%d\n",addr);
					k=atomicAdd(&(dindex_long[0]),1);
					dindex_bit[k]=addr;
					printf("%d:%d\n",k,dindex_bit[k]);
				}
			}
		}
	}
}

void cuda_malloc_cpy()
{
	int i,j,mul;
	int thread_size=3;
	int block_size=1;
	mul=(attr_total+(thread_size*block_size-1))/(thread_size*block_size);//distribution of number of tasks
	printf("mul:%d\n",mul);
	int *dindex_bit;
	int *dindex_long;
	unsigned int **dbit;
	int a[width][height];//test
	for(i=0;i<width;i++)
	{
		for(j=0;j<height;j++)
		{
			a[i][j]=0;
		}
	}
	res=hipMalloc((void **)&dindex_bit,index_size);CHECK(res);printf("\n[0] \n");
	res=hipMalloc((void **)&dindex_long,sizeof(int ));CHECK(res);printf("[1] \n");
	res=hipMalloc((void **)&dbit,bit_size);CHECK(res);printf("[2] \n");
	res=hipMemcpy(dbit,bitat,bit_size,hipMemcpyHostToDevice);CHECK(res);printf("[3] \n");
	res=hipMemcpy(dindex_long,index_long,sizeof(int ),hipMemcpyHostToDevice);CHECK(res);printf("[4] \n");
	dim3 threads(thread_size,1);
	dim3 blocks(block_size,1);
	kernel_index_bitmap<<<blocks,threads>>>(dbit,dindex_bit,dindex_long,attr_total,attr_size,height,mul);
	printf("---------------T_T-------------\n");
	res=hipMemcpy(index_bit,dindex_bit,index_size,hipMemcpyDeviceToHost);CHECK(res);printf("[5] \n");
	res=hipMemcpy(index_long,dindex_long,sizeof(int ),hipMemcpyDeviceToHost);CHECK(res);printf("[6] \n");
	res=hipMemcpy(a,dbit,bit_size,hipMemcpyDeviceToHost);CHECK(res);printf("[7] \n");
	printf("long:%d\n",index_long[0]);
	for(i=0;i<index_long[0];i++)
		printf("%d,",index_bit[i]);
	printf("\n");
	for(i=0;i<attr_total;i++)
	{
		for(j=0;j<attr_size;j++)
		{
			printf("%d,",a[i][j]);
		}
		printf("\n");
	}
	hipFree(dbit);

}

int main()
{
	get_attr_size();
	get_bitmap();
	get_total();
	get_attr();
	cuda_malloc_cpy();

	return 0;
}
